#include "hip/hip_runtime.h"
#include "Engine/FBX/CudaFiles/DDMV0.cuh"
#include "Engine/FBX/CudaFiles/FastSVD3.cuh"
#include "Engine/FBX/CudaFiles/CudaMatrixMathFunctions.cuh"
#include "Engine/Core/ErrorWarningAssert.hpp"
#include <hip/hip_runtime.h>

__global__
void ComputeV0Global(const float* allJointTransforms, int numJoints, int numControlPoints, const double* omegaMatrix, const double* controlPointsMatrix, float* outControlPointsMatrix) {
	int cpIdx = blockDim.x * blockIdx.x + threadIdx.x;
	if (cpIdx >= numControlPoints) {
		return;
	}

	double QMatrix_i[16] = {
		0.0, 0.0, 0.0, 0.0,
		0.0, 0.0, 0.0, 0.0,
		0.0, 0.0, 0.0, 0.0,
		0.0, 0.0, 0.0, 0.0
	};

	for (int jointIdx = 0; jointIdx < numJoints; jointIdx++) {
		int omegaStartIdx = (10 * numJoints) * cpIdx + jointIdx * 10;
		double symMat[16] = {
			omegaMatrix[omegaStartIdx + 0], omegaMatrix[omegaStartIdx + 1], omegaMatrix[omegaStartIdx + 2], omegaMatrix[omegaStartIdx + 3],
			omegaMatrix[omegaStartIdx + 1], omegaMatrix[omegaStartIdx + 4], omegaMatrix[omegaStartIdx + 5], omegaMatrix[omegaStartIdx + 6],
			omegaMatrix[omegaStartIdx + 2], omegaMatrix[omegaStartIdx + 5], omegaMatrix[omegaStartIdx + 7], omegaMatrix[omegaStartIdx + 8],
			omegaMatrix[omegaStartIdx + 3], omegaMatrix[omegaStartIdx + 6], omegaMatrix[omegaStartIdx + 8], omegaMatrix[omegaStartIdx + 9]
		};

		int jointTransformStartIdx = 16 * jointIdx;
		double jointTransform[16] = {
			(double)allJointTransforms[jointTransformStartIdx + 0], (double)allJointTransforms[jointTransformStartIdx + 4], (double)allJointTransforms[jointTransformStartIdx + 8],  (double)allJointTransforms[jointTransformStartIdx + 12],
			(double)allJointTransforms[jointTransformStartIdx + 1], (double)allJointTransforms[jointTransformStartIdx + 5], (double)allJointTransforms[jointTransformStartIdx + 9],  (double)allJointTransforms[jointTransformStartIdx + 13],
			(double)allJointTransforms[jointTransformStartIdx + 2], (double)allJointTransforms[jointTransformStartIdx + 6], (double)allJointTransforms[jointTransformStartIdx + 10], (double)allJointTransforms[jointTransformStartIdx + 14],
			(double)allJointTransforms[jointTransformStartIdx + 3], (double)allJointTransforms[jointTransformStartIdx + 7], (double)allJointTransforms[jointTransformStartIdx + 11], (double)allJointTransforms[jointTransformStartIdx + 15]
		};
		/*
		double jointTransform[16] = {
			(double)allJointTransforms[jointTransformStartIdx + 0], (double)allJointTransforms[jointTransformStartIdx + 1], (double)allJointTransforms[jointTransformStartIdx + 2],  (double)allJointTransforms[jointTransformStartIdx + 3],
			(double)allJointTransforms[jointTransformStartIdx + 4], (double)allJointTransforms[jointTransformStartIdx + 5], (double)allJointTransforms[jointTransformStartIdx + 6],  (double)allJointTransforms[jointTransformStartIdx + 7],
			(double)allJointTransforms[jointTransformStartIdx + 8], (double)allJointTransforms[jointTransformStartIdx + 9], (double)allJointTransforms[jointTransformStartIdx + 10], (double)allJointTransforms[jointTransformStartIdx + 11],
			(double)allJointTransforms[jointTransformStartIdx + 12], (double)allJointTransforms[jointTransformStartIdx + 13], (double)allJointTransforms[jointTransformStartIdx + 14], (double)allJointTransforms[jointTransformStartIdx + 15]
		};
		*/

		double productMat[16] = {
			0.0, 0.0, 0.0, 0.0,
			0.0, 0.0, 0.0, 0.0,
			0.0, 0.0, 0.0, 0.0,
			0.0, 0.0, 0.0, 0.0
		};

		mult4x4(jointTransform, symMat, productMat);
		add4x4(QMatrix_i, productMat, QMatrix_i);
	}

	for (int i = 0; i < 16; i++) {
		QMatrix_i[i] /= QMatrix_i[15];
	}

	double Q_i[9];
	getLeftTop3x3From4x4(QMatrix_i, Q_i);
	double q_i[3] = {QMatrix_i[3], QMatrix_i[7], QMatrix_i[11]};
	double p_i[3] = {QMatrix_i[12], QMatrix_i[13], QMatrix_i[14]};
	double q_i_mult_p_i[9];
	mult3x1With1x3(q_i, p_i, q_i_mult_p_i);
	
	double U_S_Vt[9];
	subtract3x3(Q_i, q_i_mult_p_i, U_S_Vt);

	float U[9];
	float S[9];
	float V[9];

	svd((float)U_S_Vt[0], (float)U_S_Vt[1], (float)U_S_Vt[2],
		(float)U_S_Vt[3], (float)U_S_Vt[4], (float)U_S_Vt[5],
		(float)U_S_Vt[6], (float)U_S_Vt[7], (float)U_S_Vt[8],
		U[0], U[1], U[2],
		U[3], U[4], U[5],
		U[6], U[7], U[8],
		S[0], S[1], S[2],
		S[3], S[4], S[5],
		S[6], S[7], S[8],
		V[0], V[1], V[2],
		V[3], V[4], V[5],
		V[6], V[7], V[8]
	);

	float V_t[9] = {
		V[0], V[3], V[6],
		V[1], V[4], V[7],
		V[2], V[5], V[8]
	};

	float R_i[9];
	mult3x3(U, V_t, R_i);

	float R_i_mult_p_i[3];
	float p_i_float[3] = {(float)p_i[0], (float)p_i[1], (float)p_i[2]};
	mult3x3With3x1(R_i, p_i_float, R_i_mult_p_i);
	float t_i[3];
	float q_i_float[3] = {(float)q_i[0], (float)q_i[1], (float)q_i[2]};
	subtract3x1(q_i_float, R_i_mult_p_i, t_i);

	float gamma_i[16] = {
		R_i[0], R_i[1], R_i[2], t_i[0],
		R_i[3], R_i[4], R_i[5], t_i[1],
		R_i[6], R_i[7], R_i[8], t_i[2],
		0.0f, 0.0f, 0.0f, 1.0f
	};

	/*
	float gamma_i[16] = {
		1.0f, 0.0f, 0.0f, 10.0f,
		0.0f, 1.0f, 0.0f, 10.0f,
		0.0f, 0.0f, 1.0f, 10.0f,
		0.0f, 0.0f, 0.0f, 1.0f
	};
	*/

	int controlPointsStartIdx = cpIdx * 3;
	float affineCurrentControlPoint[4] = {
		(float)controlPointsMatrix[controlPointsStartIdx], (float)controlPointsMatrix[controlPointsStartIdx + 1], (float)controlPointsMatrix[controlPointsStartIdx + 2], 1.0f
	};

	float transformedAffineCP[4];
	mult4x4With4x1(gamma_i, affineCurrentControlPoint, transformedAffineCP);

	outControlPointsMatrix[controlPointsStartIdx]	  = transformedAffineCP[0];
	outControlPointsMatrix[controlPointsStartIdx + 1] = transformedAffineCP[1];
	outControlPointsMatrix[controlPointsStartIdx + 2] = transformedAffineCP[2];
}

void ComputeV0_CUDA(const float* allJointTransforms, int numThreadGroup, int numThreadsInGroup, int numJoints, int numControlPoints, const double* omegaMatrixGPU, const double* controlPointsMatrixGPU, float* outControlPointsMatrixGPU) {
	hipEvent_t start, stop;
	float milliseconds = 0;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	// Record the start event
	hipEventRecord(start, 0);
	ComputeV0Global<<<numThreadGroup, numThreadsInGroup>>>(allJointTransforms, numJoints, numControlPoints, omegaMatrixGPU, controlPointsMatrixGPU, outControlPointsMatrixGPU);
	// Record the stop event
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);

	// Calculate the elapsed time
	hipEventElapsedTime(&milliseconds, start, stop); 
	
	DebuggerPrintf("DDMV0 Kernel execution time: %f milliseconds\n", milliseconds);

	// Clean up
	hipEventDestroy(start);
	hipEventDestroy(stop);
}
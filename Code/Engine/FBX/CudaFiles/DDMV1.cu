#include "hip/hip_runtime.h"
#include "Engine/FBX/CudaFiles/DDMV1.cuh"
#include "Engine/FBX/CudaFiles/CudaMatrixMathFunctions.cuh"
#include "Engine/Core/ErrorWarningAssert.hpp"
#include <hip/hip_runtime.h>

__global__
void ComputeV1Global(const float* allJointTransforms, int numJoints, int numControlPoints, const double* omegaMatrix, const double* controlPointsMatrix, const double* ddmV1Constants, float* outControlPointsMatrix) {
	int cpIdx = blockDim.x * blockIdx.x + threadIdx.x;
	if (cpIdx >= numControlPoints) {
		return;
	}

	double QMatrix_i[16] = {
		0.0, 0.0, 0.0, 0.0,
		0.0, 0.0, 0.0, 0.0,
		0.0, 0.0, 0.0, 0.0,
		0.0, 0.0, 0.0, 0.0
	};

	for (int jointIdx = 0; jointIdx < numJoints; jointIdx++) {
		int omegaStartIdx = (10 * numJoints) * cpIdx + jointIdx * 10;
		double symMat[16] = {
			omegaMatrix[omegaStartIdx + 0], omegaMatrix[omegaStartIdx + 1], omegaMatrix[omegaStartIdx + 2], omegaMatrix[omegaStartIdx + 3],
			omegaMatrix[omegaStartIdx + 1], omegaMatrix[omegaStartIdx + 4], omegaMatrix[omegaStartIdx + 5], omegaMatrix[omegaStartIdx + 6],
			omegaMatrix[omegaStartIdx + 2], omegaMatrix[omegaStartIdx + 5], omegaMatrix[omegaStartIdx + 7], omegaMatrix[omegaStartIdx + 8],
			omegaMatrix[omegaStartIdx + 3], omegaMatrix[omegaStartIdx + 6], omegaMatrix[omegaStartIdx + 8], omegaMatrix[omegaStartIdx + 9]
		};

		int jointTransformStartIdx = 16 * jointIdx;
		double jointTransform[16] = {
			(double)allJointTransforms[jointTransformStartIdx + 0], (double)allJointTransforms[jointTransformStartIdx + 4], (double)allJointTransforms[jointTransformStartIdx + 8],  (double)allJointTransforms[jointTransformStartIdx + 12],
			(double)allJointTransforms[jointTransformStartIdx + 1], (double)allJointTransforms[jointTransformStartIdx + 5], (double)allJointTransforms[jointTransformStartIdx + 9],  (double)allJointTransforms[jointTransformStartIdx + 13],
			(double)allJointTransforms[jointTransformStartIdx + 2], (double)allJointTransforms[jointTransformStartIdx + 6], (double)allJointTransforms[jointTransformStartIdx + 10], (double)allJointTransforms[jointTransformStartIdx + 14],
			(double)allJointTransforms[jointTransformStartIdx + 3], (double)allJointTransforms[jointTransformStartIdx + 7], (double)allJointTransforms[jointTransformStartIdx + 11], (double)allJointTransforms[jointTransformStartIdx + 15]
		};
		/*
		double jointTransform[16] = {
			(double)allJointTransforms[jointTransformStartIdx + 0], (double)allJointTransforms[jointTransformStartIdx + 1], (double)allJointTransforms[jointTransformStartIdx + 2],  (double)allJointTransforms[jointTransformStartIdx + 3],
			(double)allJointTransforms[jointTransformStartIdx + 4], (double)allJointTransforms[jointTransformStartIdx + 5], (double)allJointTransforms[jointTransformStartIdx + 6],  (double)allJointTransforms[jointTransformStartIdx + 7],
			(double)allJointTransforms[jointTransformStartIdx + 8], (double)allJointTransforms[jointTransformStartIdx + 9], (double)allJointTransforms[jointTransformStartIdx + 10], (double)allJointTransforms[jointTransformStartIdx + 11],
			(double)allJointTransforms[jointTransformStartIdx + 12], (double)allJointTransforms[jointTransformStartIdx + 13], (double)allJointTransforms[jointTransformStartIdx + 14], (double)allJointTransforms[jointTransformStartIdx + 15]
		};
		*/

		double productMat[16] = {
			0.0, 0.0, 0.0, 0.0,
			0.0, 0.0, 0.0, 0.0,
			0.0, 0.0, 0.0, 0.0,
			0.0, 0.0, 0.0, 0.0
		};

		mult4x4(jointTransform, symMat, productMat);
		add4x4(QMatrix_i, productMat, QMatrix_i);
	}

	for (int i = 0; i < 16; i++) {
		QMatrix_i[i] /= QMatrix_i[15];
	}

	double Q_i[9];
	getLeftTop3x3From4x4(QMatrix_i, Q_i);
	double q_i[3] = { QMatrix_i[3], QMatrix_i[7], QMatrix_i[11] };
	double p_i[3] = { QMatrix_i[12], QMatrix_i[13], QMatrix_i[14] };
	double q_i_mult_p_i[9];
	mult3x1With1x3(q_i, p_i, q_i_mult_p_i);

	double Q_qp[9];
	subtract3x3(Q_i, q_i_mult_p_i, Q_qp);

	double determinant = getDeterminantOf3x3(Q_qp);
	double Q_qp_t[9];
	transpose3x3(Q_qp, Q_qp_t);

	double Q_qp_t_inv[9];
	calcInvMat3x3(Q_qp_t, Q_qp_t_inv);

	double Q_qpDet_x_Q_qp_t_inv[9] = {
		determinant * Q_qp_t_inv[0], determinant * Q_qp_t_inv[1], determinant * Q_qp_t_inv[2],
		determinant * Q_qp_t_inv[3], determinant * Q_qp_t_inv[4], determinant * Q_qp_t_inv[5],
		determinant * Q_qp_t_inv[6], determinant * Q_qp_t_inv[7], determinant * Q_qp_t_inv[8]
	};

	int symMatStartIdx = 6 * cpIdx;
	double v1ConstSymMat[9] = {
		ddmV1Constants[symMatStartIdx + 0],		ddmV1Constants[symMatStartIdx + 1], ddmV1Constants[symMatStartIdx + 2],
		ddmV1Constants[symMatStartIdx + 1], ddmV1Constants[symMatStartIdx + 3], ddmV1Constants[symMatStartIdx + 4],
		ddmV1Constants[symMatStartIdx + 2], ddmV1Constants[symMatStartIdx + 4], ddmV1Constants[symMatStartIdx + 5]
	};

	double R_i[9];
	mult3x3(Q_qpDet_x_Q_qp_t_inv, v1ConstSymMat, R_i);

	double R_i_mult_p_i[3];
	mult3x3With3x1(R_i, p_i, R_i_mult_p_i);
	double t_i[3];
	subtract3x1(q_i, R_i_mult_p_i, t_i);

	double gamma_i[16] = {
		R_i[0], R_i[1], R_i[2], t_i[0],
		R_i[3], R_i[4], R_i[5], t_i[1],
		R_i[6], R_i[7], R_i[8], t_i[2],
		0.0, 0.0, 0.0, 1.0
	};

	/*
	float gamma_i[16] = {
		1.0f, 0.0f, 0.0f, 10.0f,
		0.0f, 1.0f, 0.0f, 10.0f,
		0.0f, 0.0f, 1.0f, 10.0f,
		0.0f, 0.0f, 0.0f, 1.0f
	};
	*/

	int controlPointsStartIdx = cpIdx * 3;
	double affineCurrentControlPoint[4] = {
		controlPointsMatrix[controlPointsStartIdx], controlPointsMatrix[controlPointsStartIdx + 1], controlPointsMatrix[controlPointsStartIdx + 2], 1.0
	};

	double transformedAffineCP[4];
	mult4x4With4x1(gamma_i, affineCurrentControlPoint, transformedAffineCP);

	outControlPointsMatrix[controlPointsStartIdx] = (float)transformedAffineCP[0];
	outControlPointsMatrix[controlPointsStartIdx + 1] = (float)transformedAffineCP[1];
	outControlPointsMatrix[controlPointsStartIdx + 2] = (float)transformedAffineCP[2];
}

void ComputeV1_CUDA(const float* allJointTransforms, int numThreadGroup, int numThreadsInGroup, int numJoints, int numControlPoints, const double* omegaMatrixGPU, const double* controlPointsMatrixGPU, const double* ddmV1Constants, float* outControlPointsMatrixGPU) {
	hipEvent_t start, stop;
	float milliseconds = 0;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	// Record the start event
	hipEventRecord(start, 0);
	ComputeV1Global << <numThreadGroup, numThreadsInGroup >> > (allJointTransforms, numJoints, numControlPoints, omegaMatrixGPU, controlPointsMatrixGPU, ddmV1Constants, outControlPointsMatrixGPU);
	// Record the stop event
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);

	// Calculate the elapsed time
	hipEventElapsedTime(&milliseconds, start, stop);

	DebuggerPrintf("DDMV1 Kernel execution time: %f milliseconds\n", milliseconds);

	// Clean up
	hipEventDestroy(start);
	hipEventDestroy(stop);
}